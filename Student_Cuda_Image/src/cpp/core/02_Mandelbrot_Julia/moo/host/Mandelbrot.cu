#include "hip/hip_runtime.h"
#include "Fractale.h"
#include "FractaleMath.h"
#include "MandelbrotMath.h"
#include "Mandelbrot.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;

extern __global__ void MandelbrotKernel(uchar4* ptrDevPixels, uint w, uint h,const DomaineMath &domaineMath, uint n, float t);

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, const DomaineMath& domaineMath): Fractale(grid,w,h,domaineMath){}
Mandelbrot::~Mandelbrot(){}


void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("Fractale rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    MandelbrotKernel<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t);

    Device::lastCudaError("Fractale rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }


