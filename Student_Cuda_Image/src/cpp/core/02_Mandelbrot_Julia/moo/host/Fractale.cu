#include "hip/hip_runtime.h"
#include "Fractale.h"
#include "FractaleMath.h"
#include "MandelbrotMath.h"

#include <iostream>

using std::cout;
using std::endl;


Fractale::Fractale(const Grid& grid,uint w, uint h, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid,w, h, "Fractale_Cuda_RGBA_uchar4", domaineMath), variateurAnimation(Interval<int>(20, 120), 1)
    {
    // Inputs
    this->n = n;

    // Tools
    this->t = 0;
    }

/**
 * Override
 */
void Fractale::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

__device__
void Fractale::workPixel(uchar4* ptrColorIJ, int i, int j, const DomaineMath& domaineMath, FractaleMath* ptrFractaleMath)
    {
    // (i,j) domaine ecran dans N2
    // (x,y) domaine math dans R2

    double x;
    double y;
    domaineMath.toXY(i, j, &x, &y); // fill (x,y) from (i,j)

    // float t=variateurAnimation.get();

    ptrFractaleMath->colorXY(ptrColorIJ, x, y); // in [01]
    }


Fractale::~Fractale(){}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

