#include "hip/hip_runtime.h"
#include "SphereCreator.h"
#include "AleaTools.h"


/*--------------------------------------*\
|*
 Constructeur
 *|
 \*-------------------------------------*/
SphereCreator::SphereCreator(int nbSpheres, int w, int h, int bord)
    {
// Inputs
    this->nbSpheres = nbSpheres;
    this->w = w;
    this->h = h;
    this->bord = bord;

// Outputs
    this->tabSphere = new Sphere[nbSpheres];

// required default constructeur of type : sphere(void)
    createSphere();
    }


SphereCreator::~SphereCreator()
    {
    delete[] tabSphere;
    }
/*--------------------------------------*\
|*
 Methode
 *|
 \*-------------------------------------*/

Sphere* SphereCreator::getTabSphere()
    {
    return tabSphere;
    }

/*--------------------------------------*\
|*
 Private
 *|
 \*-------------------------------------*/
/**
 * example
 */

void SphereCreator::createSphere(void)
    {
    AleaTools aleaTools = AleaTools();
    for (int i = 0; i < nbSpheres; i++)
	{
	float3 centre;
	centre.x = aleaTools.uniformeAB(0 + bord, h - bord);
	centre.y = aleaTools.uniformeAB(0 + bord, w - bord);
	centre.z = aleaTools.uniformeAB(10, 2 * w);
	float rayon = aleaTools.uniformeAB(20, w / 10);
	float hue01 = aleaTools.uniforme01();
	tabSphere[i] = Sphere(centre, rayon, hue01);
// Warning : sphere stack copier dans sphere heap,
// ok car pas ptr dans Sphere
	}
    }

