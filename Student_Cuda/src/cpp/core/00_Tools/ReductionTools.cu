#include "hip/hip_runtime.h"
#include "ReductionTools.h"
#include "cudaTools.h"
#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;

//host facultatif  __global__
bool testReductionTools(void);

__global__  static void kernelReductionTools(float* all);



__global__ void kernelReductionTools(float* all)
    {
    extern __shared__ float tabSM[];
    tabSM[Indice2D::tidLocal()] = 1;
    ReductionTools::reductionADD(tabSM, all);
    }


__host__ bool testReductionTools(void)
    {

    const int MP = Device::getMPCount();
    const int CORE_MP = Device::getCoreCountMP();
    const int DIM = 4;

    dim3 dg = dim3(MP,DIM,1);
    dim3 db = dim3(CORE_MP,DIM,1);
    Device::gridHeuristic(dg, db);

    float ptrTotal = 0;
    float* ptrDevTotal;
    const size_t S = sizeof(float);
    const size_t SIZE_SM = S * MP * CORE_MP * DIM;

    HANDLE_ERROR(hipMalloc(&ptrDevTotal, S));
    HANDLE_ERROR(hipMemset(ptrDevTotal, 0, S));  // l'interBlock

    Device::lastCudaError("kernelReductionTools (before)"); // temp debug
    kernelReductionTools<<<dg,db,SIZE_SM>>>(ptrDevTotal);  // asynchrone !!
    Device::lastCudaError("kernelReductionTools (after)"); // temp debug

    HANDLE_ERROR(hipMemcpy(&ptrTotal, ptrDevTotal, S, hipMemcpyDeviceToHost));

    cout << endl << "final value : " << ptrTotal << endl;

    HANDLE_ERROR(hipFree(ptrDevTotal));

    return true;
    }
