#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"

#include <stdio.h>



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


static __device__ void secondaire(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n)
    {
    secondaire(ptrDevV1, ptrDevV2, ptrDevW, n); // pas necessaire, just for fun
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


__device__ void secondaire(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n)
    {
    const int NB_THREAD=Indice2D::nbThread();
    const int TID=Indice2D::tid();

    // Debug, facultatif
    //si on met ca on doit mettre synchronise coté host qd on active ca les printf
    //faut tjours conditionner sinn cava etre activé par ts les threads
//    if (TID==0)
//	{
//	printf("Coucou from device tid = %d",TID); //required   Device::synchronize(); after the call of kernel
//	}

   int s = TID;
   while(s<n){
       ptrDevW[s] = ptrDevV1[s] + ptrDevV2[s];
       s += NB_THREAD;
       }
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

