#include "hip/hip_runtime.h"
#include "AddVector.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVector::AddVector(const Grid& grid,float* ptrV1, float* ptrV2, float* ptrW, int n) :
	ptrV1(ptrV1), ptrV2(ptrV2), ptrW(ptrW), n(n)
    {
    this->sizeOctet = n * sizeof(float); // octet

    // MM
	{
	// MM (malloc Device)
	    {
	    HANDLE_ERROR(hipMalloc(&ptrDevV1, sizeOctet));
	    //on a passerr l adresse de ptrDevV1 puisque la valeur du parametre change avant et aprese l appel de la fct
	    HANDLE_ERROR(hipMalloc(&ptrDevV2, sizeOctet));
	    HANDLE_ERROR(hipMalloc(&ptrDevW, sizeOctet));
	    }

	// MM (memset Device)
	    {
	    HANDLE_ERROR(hipMemset(ptrDevW, 0, sizeOctet));
	    //eviter d avoir les resultats de l execution d avant
	    }

	// MM (copy Host->Device)
	    {
	    // Pour info , ci dessous : ptrDevV1=ptrV1
	    HANDLE_ERROR(hipMemcpy(ptrDevV1, ptrV1, sizeOctet, hipMemcpyHostToDevice));
	    HANDLE_ERROR(hipMemcpy(ptrDevV2, ptrV2, sizeOctet, hipMemcpyHostToDevice));

	    }

	Device::lastCudaError("AddVector MM (end allocation)"); // temp debug, facultatif
	}

    // Grid
	{
	this->dg=grid.dg;
	this->db=grid.db;
	}
    }

AddVector::~AddVector(void)
    {
    //MM (device free)
	{
	HANDLE_ERROR(hipFree(ptrDevV1));
	HANDLE_ERROR(hipFree(ptrDevV2));
	HANDLE_ERROR(hipFree(ptrDevW));

	//Pour etre sur qu on n a pas d erreures !! a mettre tjrs
	Device::lastCudaError("AddVector MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void AddVector::run()
    {
    Device::lastCudaError("addVecteur (before)"); // temp debug
    //on n a pas le droit de mettre un attribut de type pointeur ici de facon direct ou indirect
    //on evite le goulet d etranglement entre cpu et gpu, on evite de passer l objet sphere ici au device
    addVector<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n); // assynchrone
    Device::lastCudaError("addVecteur (after)"); // temp debug

   //Ceci est une barriere exclplicite de synchronisation pour obliger le GPU d attendre que le calcul de addVector soit fini pour executer le device a gost
    Device::synchronize(); // Temp,debug, only for printf in  GPU

    // MM (Device -> Host)
	{
	HANDLE_ERROR(hipMemcpy(ptrW, ptrDevW, sizeOctet, hipMemcpyDeviceToHost)); // barriere synchronisation implicite
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
