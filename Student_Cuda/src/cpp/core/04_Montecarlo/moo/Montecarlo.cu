#include "hip/hip_runtime.h"
#include <stdio.h>

#include "Chrono.h"
#include "Device.h"
#include "Montecarlo.h"
#include "hiprand/hiprand_kernel.h"

using namespace std;

extern __global__ void setup_kernel(hiprandState* ptrDevGenerators, int deviceId, bool useClock);
extern __global__ void work_kernel(hiprandState* ptrDevGenerators, ullong* ptrDevTotal, ullong n);

Montecarlo::Montecarlo(const Grid& grid, ullong n) :
	dg(grid.dg), db(grid.db), nbFleches(n)
    {
    const size_t NB_GENERATORS = grid.threadCounts() * sizeof(hiprandState);

    this->resultat = 0;
    this->deviceId = Device::getDeviceId();
    this->ptrDevTotal = nullptr;
    this->ptrDevGenerators = nullptr;

    HANDLE_ERROR(hipMalloc(&ptrDevGenerators, NB_GENERATORS));
    HANDLE_ERROR(hipMalloc(&ptrDevTotal, sizeof(ullong)));
    Device::lastCudaError("Montecarlo MM (end allocation)");
    }

Montecarlo::~Montecarlo()
    {
    HANDLE_ERROR(hipFree(ptrDevGenerators));
    HANDLE_ERROR(hipFree(ptrDevTotal));
    Device::lastCudaError("Montecarlo MM (end deallocation)");
    }

void Montecarlo::setDeviceId(int id)
    {
    // Permet de changer le deviceId manuellement. Pas utilisé.
    this->deviceId = id;
    }

void Montecarlo::run()
    {
    const size_t SIZE_SM = db.x * sizeof(ullong);
    HANDLE_ERROR(hipMemset(this->ptrDevTotal, 0, sizeof(ullong)));

#ifdef MONTE_DEBUG
    Chrono chrono1;
#endif
    setup_kernel<<<dg,db>>>(this->ptrDevGenerators, this->deviceId, clock);
#ifdef MONTE_DEBUG
    Device::synchronize();
    chrono1.stop();

    Chrono chrono2;
#endif
    work_kernel<<<dg,db,SIZE_SM>>>(this->ptrDevGenerators, this->ptrDevTotal, this->nbFleches);
#ifdef MONTE_DEBUG
    Device::synchronize();
    chrono2.stop();

    double chronoTotal = chrono1.getDeltaTime() + chrono2.getDeltaTime();
    cout << "Temps Générateurs: " << chrono1.getDeltaTime() << " sec." << endl;
    cout << "Temps Calculs: " << chrono2.getDeltaTime() << " sec." << endl;
    cout << "Temps Total: " << chronoTotal << " sec." << endl;
#endif

    ullong temp;
    HANDLE_ERROR(hipMemcpy(&temp, this->ptrDevTotal, sizeof(ullong), hipMemcpyDeviceToHost));
    this->resultat = (ullong)temp;
    }
