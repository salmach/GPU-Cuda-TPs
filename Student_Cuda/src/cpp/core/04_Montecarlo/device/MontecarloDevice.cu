#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Indice2D.h>
#include <chrono>

#include "hiprand/hiprand_kernel.h"
#include "Calibreur_GPU.h"
#include "Grid.h"

#include "Montecarlo.h"
#include "ReductionTools.h"

using namespace gpu;

__global__ void setup_kernel(hiprandState* ptrDevGenerators, int deviceId);
__global__ void work_kernel(hiprandState* ptrDevGenerators, ullong* ptrDevTotal, ullong n);
static __device__ bool isInside(float x, float y, float m, float(*fct)(float));
static __device__ float f(float x);
static __device__ float g(float x);

// Make this function compatible in mono / multi GPU
__global__ void setup_kernel(hiprandState* ptrDevGenerators, int deviceId, bool useClock)
    {
    const int TID = Indice2D::tid();

    int deltaSeed = deviceId * INT_MAX;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;

    int seed = (useClock) ? clock64() + deltaSeed : 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;

    hiprand_init(seed, sequenceNumber, offset, &ptrDevGenerators[TID]);
    }

__global__ void work_kernel(hiprandState* ptrDevGenerators, ullong* ptrDevTotal, ullong n)
    {
    extern __shared__ ullong tabSM[];

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    hiprandState localGenerator = ptrDevGenerators[TID];

    ullong somme = 0;
    ullong s = TID;

    // ATTENTION: dépend de la fonction utilisée dans isInside()
    // m >= y Max de la fonction
    float m = 1;

    while(s<n)
	{
	// Call isInside() with f() as testing function.
	// Can be replace by g() or any function with the same signature
	somme += isInside(hiprand_uniform(&localGenerator), hiprand_uniform(&localGenerator), m, f);
	s += NB_THREAD;
	}

    tabSM[Indice2D::tidLocal()] = somme;
    __syncthreads();

    ReductionTools::reductionADD(tabSM, ptrDevTotal);
    }

// Fonction déterminant si la fleche est dans la fonction
__device__ bool isInside(float x, float y, float m, float(*fct)(float))
    {
    	Calibreur<float> calY(Interval<float>(0,1), Interval<float>(0, m));
    	calY.calibrer(y);
	return (y <= fct(x)) ? 1 : 0;
    }

// Fonction pour x de -1 à 1
__device__ float f(float x)
    {
	Calibreur<float> calX(Interval<float>(0,1), Interval<float>(-1, 1));
    	calX.calibrer(x);
    	return sqrt(1 - x * x);
    }

// Fonction pour x de 0 à 1
__device__ float g(float x)
    {
	Calibreur<float> calX(Interval<float>(0,1), Interval<float>(0, 1));
    	calX.calibrer(x);
    	return 1 / (1 + x * x);
    }
